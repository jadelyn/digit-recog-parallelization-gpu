// launch the gpu computation

#include<hip/hip_runtime.h>
#include<cutil.h>
#include<sys/time.h>
#include<time.h>
#include "launch_gpu.h"

extern "C" float* launch_gpu_computation(float *image, int i_width, int i_height, float *temp, int t_width) {
    float* gpu_image;
    hipMalloc(&gpu_image, sizeof(float)*i_width*i_height);
    hipMemcpy(gpu_image, image, i_width*i_height*sizeof(float), hipMemcpyHostToDevice);

    float* gpu_temp;
    hipMalloc(&gpu_temp, sizeof(float)*t_width*t_width);
    hipMemcpy(gpu_temp, temp, t_width*t_width*sizeof(float), hipMemcpyHostToDevice);

    double seconds;
    struct timeval start, end;
    gettimeofday(&start, NULL);
    float distance_computed = calc_min_dist(gpu_image, i_width, i_height, gpu_temp, t_width);
    gettimeofday(&end, NULL);
    seconds = (end.tv_sec - start.tv_sec) + 1.0e-6 * (end.tv_usec - start.tv_usec);

    hipFree(gpu_image);
    hipFree(gpu_temp);

    float* retVal = (float*)malloc(sizeof(float)*2);
    retVal[0] = (float)seconds;
    retVal[1] = distance_computed;
    return retVal;
}


extern "C" float just_get_dist_gpu(float *image, int i_width, int i_height, float *temp, int t_width) {
    float* res;
    res = launch_gpu_computation(image, i_width, i_height, temp, t_width);
    float retVal = res[1];
    free(res);
    return retVal;
}
