/*
 * Proj 3-2 SKELETON
 */

#include <float.h>
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cutil.h>
#include "utils.h"

/* Does a horizontal flip of the array arr */
void flip_horizontal(float *arr, int width) {
    /* YOU MAY WISH TO IMPLEMENT THIS */

}

/* Transposes the square array ARR. */
void transpose(float *arr, int width) {
    /* YOU MAY WISH TO IMPLEMENT THIS */

}

/* Rotates the square array ARR by 90 degrees counterclockwise. */
void rotate_ccw_90(float *arr, int width) {
    /* YOU MAY WISH TO IMPLEMENT THIS */

}

/* Returns the squared Euclidean distance between TEMPLATE and IMAGE. The size of IMAGE
 * is I_WIDTH * I_HEIGHT, while TEMPLATE is square with side length T_WIDTH. The template
 * image should be flipped, rotated, and translated across IMAGE.
 */
float calc_min_dist(float *image, int i_width, int i_height, float *temp, int t_width) {
    // float* image and float* temp are pointers to GPU addressible memory
    // You MAY NOT copy this data back to CPU addressible memory and you MAY 
    // NOT perform any computation using values from image or temp on the CPU.

    /* YOUR CODE HERE */

    return 0;
}
